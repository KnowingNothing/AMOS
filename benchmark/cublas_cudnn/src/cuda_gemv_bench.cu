#include <chrono>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <memory>
#include <sstream>
#include <stdexcept>
#include <tuple>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "tensor.h"


// y = alpha * op ( A ) x + beta * y
// if not transpose:
// A shape: m X n, x: n, y: m

// Vector saves m, n, trans
std::vector<std::tuple<int, int, bool>> inference_server_set = {
    std::make_tuple(1024, 256, false),
    std::make_tuple(512, 1024, false)
    };


template <typename T1, typename T2>
int time_gemv(Tensor<T1> A, Tensor<T1> X, Tensor<T2> Y, int m, int n,
              bool trans, hipblasHandle_t cublas_handle) {
  float alpha = 1.0;
  float beta = 1.0;

  int numRepeats = 6;
  hipblasStatus_t stat;

  stat =
      hipblasSgemv(cublas_handle, trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   m, n, &alpha, A.begin(), A.dims()[0],
                   X.begin(), 1, &beta,
                   Y.begin(), 1);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("Sgemv failed");
  }

  hipDeviceSynchronize();

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < numRepeats; ++i) {
    stat = hipblasSgemv(cublas_handle, trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                        m, n, &alpha, A.begin(), A.dims()[0],
                        X.begin(), 1, &beta,
                        Y.begin(), 1);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
      throw std::runtime_error("Sgemv failed");
    }
  }
  hipDeviceSynchronize();

  auto end = std::chrono::steady_clock::now();

  return static_cast<int>(
      std::chrono::duration<double, std::micro>(end - start).count() /
      numRepeats);
}

int main(int argc, char **argv) {
  // Get Device Number
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  for (int dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;

    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cout << "CUBLAS init failed" << std::endl;
    }

    std::cout << "m,n,trans,fp32 time (usec)" << std::endl;

    for (const auto &problem : inference_server_set) {
      int m, n;
      bool trans;
      std::tie(m, n, trans) = problem;
      int time_ms;

      std::cout << m << ",";
      std::cout << n << ",";
      std::cout << trans ? "t" : "n" ;

      // set cublas to not use tensor core
      status = hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS math mode failed" << std::endl;
      }

      // fp32 benchmark
      {
        auto a = rand<float>({trans? n : m, trans? m : n}, curand_gen);
        auto x = rand<float>({trans ? m : n}, curand_gen);
        auto y = zeros<float>({trans ? n : m});
        time_ms =
            time_gemv<float, float>(a, x, y, m, n, trans, cublas_handle);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      std::cout << std::endl;

    }

    hipblasDestroy(cublas_handle);
    hiprandDestroyGenerator(curand_gen);
  }

  return 0;
}
