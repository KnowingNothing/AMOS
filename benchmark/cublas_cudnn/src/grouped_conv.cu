#include <chrono>
#include <iomanip>
#include <memory>
#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "cudnn_helper.h"
#include "tensor.h"
#include "configs.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

// Vector saves b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew
std::vector<std::tuple<unsigned int, unsigned int, unsigned int, unsigned int,
                       unsigned int, unsigned int, unsigned int, unsigned int,
                       unsigned int, unsigned int, unsigned int, unsigned int>>
    inference_server_set = {
        std::make_tuple(100, 50, 3, 64, 32, 32, 3, 3, 0, 0, 1, 1)
      };

/*
Usage:

The default precision is set based on the architecture and mode.

By default, the program runs the benchmark in training mode.

bin/conv_bench

To run inference mode, use the following command:

bin/conv_bench inference


To change the precision for training/inference, use:

bin/conv_bench train <precision>
bin/conv_bench inference <precision>

Supported precision types:

For Maxwell GPUS:
float for training and inference

For Pascal GPUS:
float, half for training
float, half, int8 for inference

*/

// T1 is used as the data type for inputs, weights and outputs.
// T2 is used to describe the compute precision. This is used in inference mode
// in the INT8_CONFIG
template <typename T1, typename T2> class cudnnCNN {
  TensorDescriptor4d<T1> x_desc_;
  TensorDescriptor4d<T1> h_desc_;

  FilterDescriptor4d<T1> w_desc_;

  std::vector<int> output_dims_;
  int num_repeats_;

  size_t fwd_workspace_size_;
  Tensor<float> fwd_workspace_;
  hipdnnConvolutionFwdAlgo_t fwd_algo_;

  const T2 alpha_ = 1.f;
  const T2 beta_ = 0.f;

  ConvolutionDescriptor<T2> conv_desc_;
  CudnnHandle cudnn_handle_;

  public:
  cudnnCNN(int b_j, int b_i, int c_in, int c_out, int h, int w,
           int kh, int kw, int padh, int padw, int strideh, int stridew,
           bool use_tensor_core)
      : cudnn_handle_(), conv_desc_(padh, padw, strideh, stridew) {
    int outh, outw, outc, outn;

    CHECK_CUDNN_ERROR(hipdnnSetConvolutionGroupCount(conv_desc_.desc(), b_i));

    hipdnnTensorFormat_t format;
    // For int8 inference, the supported format is NHWC
    if (std::is_same<T1, uint8_t>::value) {
      format = HIPDNN_TENSOR_NHWC;
    } else {
      format = HIPDNN_TENSOR_NCHW;
    }

    x_desc_ = TensorDescriptor4d<T1>(format, b_j, b_i * c_in, h, w);
    w_desc_ = FilterDescriptor4d<T1>(format, b_i * c_out, c_in, kh, kw);

    hipdnnMathType_t algo =
        use_tensor_core ? HIPDNN_TENSOR_OP_MATH : HIPDNN_DEFAULT_MATH;

    hipdnnSetConvolutionMathType(conv_desc_.desc(), algo);
    // Get output dimensions
    CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc_.desc(), x_desc_.desc(), w_desc_.desc(), &outn, &outc,
        &outh, &outw));

    h_desc_ = TensorDescriptor4d<T1>(format, outn, outc, outh, outw);

    output_dims_ = {outw, outh, outc, outn};

    // Pick forward convolution algorithm
    hipdnnConvolutionFwdAlgoPerf_t fwd_perf;
    int ret_count;

    CHECK_CUDNN_ERROR(hipdnnFindConvolutionForwardAlgorithm(
        cudnn_handle_.handle(), x_desc_.desc(), w_desc_.desc(),
        conv_desc_.desc(), h_desc_.desc(), 1, &ret_count, &fwd_perf));
    fwd_algo_ = fwd_perf.algo;

    if (use_tensor_core) {
      // Tensor Op math only supports IMPLICIT_PRECOMP_GEMM algorithm
      fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    }
    if (std::is_same<T1, uint8_t>::value) {
      // Note: cudnn workspace size function doesn't work for INT8_CONFIG
      fwd_workspace_size_ = 1073741824;
    } else {
      // Set fwd workspace size
      CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(
          cudnn_handle_.handle(), x_desc_.desc(), w_desc_.desc(),
          conv_desc_.desc(), h_desc_.desc(), fwd_algo_, &fwd_workspace_size_));
    }

    fwd_workspace_ = zeros<float>(std::vector<int>{
        static_cast<int>(fwd_workspace_size_ / sizeof(float)), 1});
  }

  std::vector<int> get_output_dims() { return output_dims_; }

  std::string get_fwd_algo_string() {
    if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM)
      return "IMPLICIT_GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM)
      return "IMPLICIT_PRECOMP_GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_GEMM)
      return "GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT)
      return "DIRECT";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT)
      return "FFT";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING)
      return "FFT_TILING";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD)
      return "WINOGRAD";
#if CUDNN_MAJOR >= 6
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED)
      return "WINOGRAD_NONFUSED";
#endif
    else {
      std::stringstream ss;
      ss << "Illegal algorithm passed to get_fwd_algo_string. Algo: "
         << fwd_algo_ << std::endl;
      throw std::runtime_error(ss.str());
    }
  }

  void forward(Tensor<T1> x, Tensor<T1> filter, Tensor<T1> h) {
    // Convolution forward.
    CHECK_CUDNN_ERROR(hipdnnConvolutionForward(
        cudnn_handle_.handle(), &alpha_, x_desc_.desc(), x.begin(),
        w_desc_.desc(), filter.begin(), conv_desc_.desc(), fwd_algo_,
        fwd_workspace_.begin(), fwd_workspace_size_, &beta_, h_desc_.desc(),
        h.begin()));
  }
};

template <typename T1, typename T2>
int time_cnn(int b_j, int b_i, int c_in, int c_out, int h, int w,
             int kh, int kw, int padh, int padw, int strideh, int stridew,
             int num_repeats, hiprandGenerator_t curand_gen, bool use_tensor_core) {

  cudnnCNN<T1, T2> cnn(b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
                       use_tensor_core);

  // Allocate memory for filter
  auto filter = rand<T1>(std::vector<int>{kw, kh, c_in, b_i * c_out}, curand_gen);

  // Allocate memory for input
  auto input = rand<T1>(std::vector<int>{w, h, b_i * c_in, b_j}, curand_gen);

  // Allocate memory for output tensor
  auto output = zeros<T1>(cnn.get_output_dims());

  // Warm up
  cnn.forward(input, filter, output);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < num_repeats; ++i) {
    cnn.forward(input, filter, output);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  int fwd_time = static_cast<int>(
      std::chrono::duration<double, std::micro>(end - start).count() /
      num_repeats);

  return fwd_time;
}

int main(int argc, char **argv) {
  int num_repeats = 20;

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  for (int dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;

    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    std::cout
        << "b_j,b_i,c_in,c_out,h,w,kh,kw,padh,padw,strideh,stridew,fp32 time "
           "(usec),fp16 time (usec),int8 time "
           "(usec),fp16 tensor core time (usec),int8 tensor core time (usec)"
        << std::endl;

    int pad_kernels_count = 0;

    for (const auto &problem : inference_server_set) {
      int b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew;
      std::tie(b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew) = problem;
      // n = batch;
      int fwd_time;

      std::cout << b_j << ",";
      std::cout << b_i << ",";
      std::cout << c_in << ",";
      std::cout << c_out << ",";
      std::cout << h << ",";
      std::cout << w << ",";
      std::cout << kh << ",";
      std::cout << kw << ",";
      std::cout << padh << ",";
      std::cout << padw << ",";
      std::cout << strideh << ",";
      std::cout << stridew;

      // fp32 benchmark
      {

        fwd_time = time_cnn<float, float>(
            b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
            num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // fp16 benchmark
      {

        fwd_time = time_cnn<uint16_t, uint16_t>(
            b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
            num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // int8 benchmark
      {
        int pad_value;

        pad_value = 4;
        if (c_in % pad_value || w % pad_value || h % pad_value) {
          pad_kernels_count++;
          pad_dim(c_in, pad_value);
          pad_dim(h, pad_value);
          pad_dim(w, pad_value);
        }
        fwd_time = time_cnn<uint8_t, int>(
            b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
            num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // fp16 tensor core benchmark
      {

        fwd_time = time_cnn<uint16_t, uint16_t>(
            b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
            num_repeats, curand_gen, true);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // int8 tensor core benchmark
      {
        int pad_value;

        pad_value = 4;
        if (c_in % pad_value || w % pad_value || h % pad_value) {
          pad_kernels_count++;
          pad_dim(c_in, pad_value);
          pad_dim(h, pad_value);
          pad_dim(w, pad_value);
        }
        fwd_time = time_cnn<uint8_t, int>(
            b_j, b_i, c_in, c_out, h, w, kh, kw, padh, padw, strideh, stridew,
            num_repeats, curand_gen, true);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      std::cout << std::endl;
    }

    // Destroy all the handles
    hiprandDestroyGenerator(curand_gen);
  }

  return 0;
}