#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <memory>
#include <sstream>
#include <stdexcept>
#include <tuple>
#include <vector>
#include <algorithm>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "tensor.h"
#include "configs.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

template <typename T1, typename T2>
int time_gemm(Tensor<T1> A, Tensor<T1> B, Tensor<T2> C, bool a_t, bool b_t,
              hipblasHandle_t cublas_handle, bool use_tensor_core) {
  const T2 alpha = 1.f;
  const T2 beta = 0.f;

  int m = C.dims()[0];
  int k = a_t ? A.dims()[0] : A.dims()[1];
  int n = C.dims()[1];

  int numRepeats = 20;
  hipblasStatus_t stat;

  hipDataType A_type = HIP_R_32F;
  hipDataType B_type = HIP_R_32F;
  hipDataType C_type = HIP_R_32F;
  hipDataType compute_type = HIP_R_32F;
  hipblasGemmAlgo_t algo;

  if (std::is_same<T1, uint16_t>::value) {
    A_type = HIP_R_16F;
    B_type = HIP_R_16F;
    C_type = HIP_R_16F;
    compute_type = HIP_R_16F;
  }

  if (std::is_same<T1, uint8_t>::value) {
    A_type = HIP_R_8I;
    B_type = HIP_R_8I;
    C_type = HIP_R_32I;
    compute_type = HIP_R_32I;
  }

  algo = use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;

  stat =
      hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha, A.begin(),
                   A_type, A.dims()[0], B.begin(), B_type, B.dims()[0], &beta,
                   C.begin(), C_type, C.dims()[0], compute_type, algo);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("sgemm failed");
  }

  hipDeviceSynchronize();

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < numRepeats; ++i) {
    stat = hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                        b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha,
                        A.begin(), A_type, A.dims()[0], B.begin(), B_type,
                        B.dims()[0], &beta, C.begin(), C_type, C.dims()[0],
                        compute_type, algo);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
      throw std::runtime_error("sgemm failed");
    }
  }
  hipDeviceSynchronize();

  auto end = std::chrono::steady_clock::now();

  return static_cast<int>(
      std::chrono::duration<double, std::micro>(end - start).count() /
      numRepeats);
}

template<typename T1>
void rand_matrix(T1 *ptr, int size) {
  for (int i = 0; i < size; ++i)
    ptr[i] = i;
}

template <typename T1, typename T2>
bool check_gemm(Tensor<T1> A, Tensor<T1> B, Tensor<T2> C, bool a_t, bool b_t,
              hipblasHandle_t cublas_handle, bool use_tensor_core) {

  const T2 alpha = 1.f;
  const T2 beta = 0.f;

  int m = C.dims()[0];
  int k = a_t ? A.dims()[0] : A.dims()[1];
  int n = C.dims()[1];

  int lda = A.dims()[0], ldb = B.dims()[0], ldc = C.dims()[0];
  T1 *A_host, *B_host;
  T2 *C_host, *C_reference;

  A_host = (T1 *)malloc(A.size() * sizeof(T1));
  rand_matrix(A_host, A.size());
  hipMemcpy(A.begin(), A_host, A.size() * sizeof(T1), hipMemcpyHostToDevice);
  B_host = (T1 *)malloc(B.size() * sizeof(T1));
  rand_matrix(B_host, B.size());
  hipMemcpy(B.begin(), B_host, B.size() * sizeof(T1), hipMemcpyHostToDevice);
  C_reference = (T2 *)malloc(C.size() * sizeof(T2));
  hipMemcpy(C_reference, C.begin(), C.size() * sizeof(T2), hipMemcpyDeviceToHost);

  // int numRepeats = 6;
  hipblasStatus_t stat;

  hipDataType A_type = HIP_R_32F;
  hipDataType B_type = HIP_R_32F;
  hipDataType C_type = HIP_R_32F;
  hipDataType compute_type = HIP_R_32F;
  hipblasGemmAlgo_t algo;

  if (std::is_same<T1, uint16_t>::value) {
    A_type = HIP_R_16F;
    B_type = HIP_R_16F;
    C_type = HIP_R_16F;
    compute_type = HIP_R_16F;
  }

  if (std::is_same<T1, uint8_t>::value) {
    A_type = HIP_R_8I;
    B_type = HIP_R_8I;
    C_type = HIP_R_32I;
    compute_type = HIP_R_32I;
  }

  algo = use_tensor_core ? CUBLAS_GEMM_DFALT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT;

  stat =
      hipblasGemmEx(cublas_handle, a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha, A.begin(),
                   A_type, A.dims()[0], B.begin(), B_type, B.dims()[0], &beta,
                   C.begin(), C_type, C.dims()[0], compute_type, algo);
  
  C_host = (T2 *)malloc(C.size() * sizeof(T2));
  hipMemcpy(C_host, C.begin(), C.size() * sizeof(T2), hipMemcpyDeviceToHost);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("sgemm failed");
  }
  
  for (int i = 0; i < m; ++i)
  for (int j = 0; j < n; ++j) {
    T2 accumulator = 0;
    for (int z = 0; z < k; ++z) {
      accumulator += A_host[i * k + z] * B_host[z * n + j];
    }
    C_reference[i * n + j] = alpha * accumulator + beta * C_reference[i * n + j];
  }

  float rtol = 1e-2, atol = 1e-2;
  for (int i = 0; i < m; ++i)
  for (int j = 0; j < n; ++j) {
    float C_diff = std::abs(C_host[i * n + j] - C_reference[i * n + j]);
    float C_min = std::min(std::abs(C_host[i * n + j]), std::abs(C_reference[i * n + j]));
    if (!(C_diff <= atol && C_diff / C_min <= rtol)) {
      std::cout << "\n(" << i << "," << j << "): atol=" << C_diff << ", rtol=" << C_diff/C_min << std::endl;
      std::cout << "-------" << std::endl;
      std::cout << C_host[i * n + j] << "," << C_reference[i * n + j] << std::endl;
      std ::cout << A.dims()[0] << "," << A.dims()[1] << std::endl;
      std ::cout << B.dims()[0] << "," << B.dims()[1] << std::endl;
      std ::cout << C.dims()[0] << "," << C.dims()[1] << std::endl;
      return false;
    }
  }
  return true;
}

int main(int argc, char **argv) {
  // Get Device Number
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  int inference = 1;
  if (argc > 1) {
    std::string inf = "inference";
    inference = argv[1] == inf ? 1 : 0;
  }

  if (inference) {
    std::cout << "Running inference benchmark " << std::endl;
  } else {
    std::cout << "Running training benchmark " << std::endl;
  }

  for (int dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;

    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cout << "CUBLAS init failed" << std::endl;
    }

    std::cout
        << "m,n,k,a_t,b_t,fp32 time (usec),fp16 time (usec),int8 time "
           "(usec),fp16 tensor core time (usec),int8 tensor core time (usec)"
        << std::endl;

    int pad_kernels_count = 0;


    //--------------------------------------------
    int m = 32, n = 32, k = 16;
    bool a_t = false, b_t = false;
    auto a = rand<float>({a_t ? k : m, a_t ? m : k}, curand_gen);
    auto b = rand<float>({b_t ? n : k, b_t ? k : n}, curand_gen);
    auto c = zeros<float>({m, n});
    if (!check_gemm<float, float>(a, b, c, a_t, b_t, cublas_handle, false)) {
      std::cout << "gemm check fail!" << std::endl;
      return 0;
    }
    else std::cout << "gemm check pass!" << std::endl;

    //--------------------------------------------

    for (const auto &problem : gemm_set) {
      int m, n, k;
      bool a_t, b_t;
      std::tie(m, n, k, a_t, b_t) = problem;
      int time_ms;

      std::cout << m << ",";
      std::cout << n << ",";
      std::cout << k << ",";
      std::cout << "n"
                << ",";
      std::cout << "n";

      // set cublas to not use tensor core
      status = hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS math mode failed" << std::endl;
      }


      // fp32 benchmark
      {
        auto a = rand<float>({a_t ? k : m, a_t ? m : k}, curand_gen);
        auto b = rand<float>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<float>({m, n});
        time_ms =
            time_gemm<float, float>(a, b, c, a_t, b_t, cublas_handle, false);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      // fp16 benchmark
      {
        auto a = rand<uint16_t>({a_t ? k : m, a_t ? m : k}, curand_gen);
        auto b = rand<uint16_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<uint16_t>({m, n});
        time_ms = time_gemm<uint16_t, uint16_t>(a, b, c, a_t, b_t,
                                                cublas_handle, false);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      // int8 benchmark
      {
        int pad_m;
        pad_m = m;
        if (pad_m % 4) {
          pad_kernels_count++;
          pad_dim(pad_m, 4);
        }

        auto a = rand<uint8_t>({a_t ? k : pad_m, a_t ? pad_m : k}, curand_gen);
        auto b = rand<uint8_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<int>({pad_m, n});
        time_ms =
            time_gemm<uint8_t, int>(a, b, c, a_t, b_t, cublas_handle, false);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      // set cublas to use tensor core
      status = hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
      if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS math mode failed" << std::endl;
      }

      // fp16 tensor core benchmark
      {
        auto a = rand<uint16_t>({a_t ? k : m, a_t ? m : k}, curand_gen);
        auto b = rand<uint16_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<uint16_t>({m, n});
        time_ms = time_gemm<uint16_t, uint16_t>(a, b, c, a_t, b_t,
                                                cublas_handle, true);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      // int8 tensor core benchmark
      {
        int pad_m;
        pad_m = m;
        if (pad_m % 4) {
          pad_kernels_count++;
          pad_dim(pad_m, 4);
        }

        auto a = rand<uint8_t>({a_t ? k : pad_m, a_t ? pad_m : k}, curand_gen);
        auto b = rand<uint8_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<int>({pad_m, n});
        time_ms =
            time_gemm<uint8_t, int>(a, b, c, a_t, b_t, cublas_handle, true);
        std::cout << "," << std::setprecision(6) << time_ms;
      }

      // std::stringstream ss;
      // ss << "Unsupported precision requested. Precision: " << precision << "
      // Inference: " << inference;

      std::cout << std::endl;
    }

    hipblasDestroy(cublas_handle);
    hiprandDestroyGenerator(curand_gen);
  }

  return 0;
}
