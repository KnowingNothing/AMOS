#include <chrono>
#include <iomanip>
#include <memory>
#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "cudnn_helper.h"
#include "tensor.h"
#include "configs.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

// T1 is used as the data type for inputs, weights and outputs.
// T2 is used to describe the compute precision. This is used in inference mode
// in the INT8_CONFIG
template <typename T1, typename T2> class cudnnCNN {
  TensorDescriptorNd<T1> x_desc_;
  TensorDescriptorNd<T1> h_desc_;
  // TensorDescriptor4d<T1> x_desc_;
  // TensorDescriptor4d<T1> h_desc_;

  FilterDescriptorNd<T1> w_desc_;

  std::vector<int> output_dims_;
  int num_repeats_;

  size_t fwd_workspace_size_;
  Tensor<float> fwd_workspace_;
  hipdnnConvolutionFwdAlgo_t fwd_algo_;

  const float alpha_ = 1.f;
  const float beta_ = 0.f;

  ConvolutionDescriptor<T2> conv_desc_;
  CudnnHandle cudnn_handle_;

public:
  cudnnCNN(int d, int w, int h, int c, int n, int k, int kernel_d, int r, int s, int pad_d, int pad_w,
           int pad_h, int dstride, int wstride, int hstride, bool use_tensor_core)
      : cudnn_handle_(), conv_desc_(pad_d, pad_h, pad_w, dstride, hstride, wstride) {

    hipdnnTensorFormat_t format;
    // For int8 inference, the supported format is NHWC
    if (std::is_same<T1, uint8_t>::value) {
      format = HIPDNN_TENSOR_NHWC;
    } else {
      format = HIPDNN_TENSOR_NCHW;
    }

    // x_desc_ = TensorDescriptor4d<T1>(format, n, c, h, w);
    // w_desc_ = FilterDescriptor4d<T1>(format, k, c, r, s);
    x_desc_ = TensorDescriptorNd<T1>(std::vector<int>{n, c, d, h, w});
    w_desc_ = FilterDescriptorNd<T1>(format, std::vector<int>{k, c, r, s, kernel_d});
    //outputDim = 1 + ( inputDim + 2*pad - (((filterDim-1)*dilation)+1) )/convolutionStride
    int tensorOuputDimA[] = {n, k, 1+(d+2*pad_d-(((kernel_d-1)*1)+1)) / dstride,
                                   1+(h+2*pad_h-(((     r  -1)*1)+1)) / hstride,
                                   1+(w+2*pad_w-(((     s  -1)*1)+1)) / wstride};

    hipdnnMathType_t algo =
        use_tensor_core ? HIPDNN_TENSOR_OP_MATH : HIPDNN_DEFAULT_MATH;

    hipdnnSetConvolutionMathType(conv_desc_.desc(), algo);
    // Get output dimensions
    // CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
    //     conv_desc_.desc(), x_desc_.desc(), w_desc_.desc(), &out_n, &out_c,
    //     &out_h, &out_w));
    
    // CHECK_CUDNN_ERROR(cudnnGetConvolutionNdForwardOutputDim(
    //       conv_desc_.desc(), x_desc_.desc(), w_desc_.desc(), 5,
    //       tensorOuputDimA
    // ));

    // h_desc_ = TensorDescriptor4d<T1>(format, out_n, out_c, out_h, out_w);
    h_desc_ = TensorDescriptorNd<T1>(std::vector<int>{tensorOuputDimA[0], tensorOuputDimA[1], 
                tensorOuputDimA[2], tensorOuputDimA[3], tensorOuputDimA[4]});

    output_dims_ = {tensorOuputDimA[0], tensorOuputDimA[1], tensorOuputDimA[2], tensorOuputDimA[3], tensorOuputDimA[4]};

    // Pick forward convolution algorithm
    hipdnnConvolutionFwdAlgoPerf_t fwd_perf;
    int ret_count;

    if (std::is_same<T1, uint8_t>::value) {
      // Note: cuDNN only supports IMPLICIT_PRECOMP_GEMM for int8 data type.
      fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    } else {
      CHECK_CUDNN_ERROR(hipdnnFindConvolutionForwardAlgorithm(
          cudnn_handle_.handle(), x_desc_.desc(), w_desc_.desc(),
          conv_desc_.desc(), h_desc_.desc(), 1, &ret_count, &fwd_perf));
      fwd_algo_ = fwd_perf.algo;
    }

    if (use_tensor_core) {
      // Tensor Op math only supports IMPLICIT_PRECOMP_GEMM algorithm
      fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    }

    fwd_workspace_size_ = 1073741824;
    // if (std::is_same<T1, uint8_t>::value) {
    //   // Note: cudnn workspace size function doesn't work for INT8_CONFIG
    //   fwd_workspace_size_ = 1073741824;
    // } else {
    //   // Set fwd workspace size
    //   assert(cudnn_handle_.handle() != NULL);
    //   assert(x_desc_.desc() != NULL);
    //   assert(w_desc_.desc() != NULL);
    //   assert(conv_desc_.desc() != NULL);
    //   assert(h_desc_.desc() != NULL);
    //   CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(
    //       cudnn_handle_.handle(), x_desc_.desc(), w_desc_.desc(),
    //       conv_desc_.desc(), h_desc_.desc(), fwd_algo_, &fwd_workspace_size_));
    // }

    fwd_workspace_ = zeros<float>(std::vector<int>{
        static_cast<int>(fwd_workspace_size_ / sizeof(float)), 1});
  }

  std::vector<int> get_output_dims() { return output_dims_; }

  std::string get_fwd_algo_string() {
    if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM)
      return "IMPLICIT_GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM)
      return "IMPLICIT_PRECOMP_GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_GEMM)
      return "GEMM";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT)
      return "DIRECT";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT)
      return "FFT";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING)
      return "FFT_TILING";
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD)
      return "WINOGRAD";
#if CUDNN_MAJOR >= 6
    else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED)
      return "WINOGRAD_NONFUSED";
#endif
    else {
      std::stringstream ss;
      ss << "Illegal algorithm passed to get_fwd_algo_string. Algo: "
         << fwd_algo_ << std::endl;
      throw std::runtime_error(ss.str());
    }
  }

  void forward(Tensor<T1> x, Tensor<T1> filter, Tensor<T1> h) {
    // Convolution forward.
    CHECK_CUDNN_ERROR(hipdnnConvolutionForward(
        cudnn_handle_.handle(), &alpha_, x_desc_.desc(), x.begin(),
        w_desc_.desc(), filter.begin(), conv_desc_.desc(), fwd_algo_,
        fwd_workspace_.begin(), fwd_workspace_size_, &beta_, h_desc_.desc(),
        h.begin()));
  }
};

template <typename T1, typename T2>
int time_cnn(int k, int c, int r, int s, int n, int d, int kernel_d, int h, int w, int pad_d, int pad_h,
             int pad_w, int dstride, int hstride, int wstride, int num_repeats,
             hiprandGenerator_t curand_gen, bool use_tensor_core) {

  cudnnCNN<T1, T2> cnn(d, w, h, c, n, k, kernel_d, r, s, pad_d, pad_w, pad_h, dstride, wstride, hstride,
                       use_tensor_core);

  // Allocate memory for filter
  // auto filter = rand<T1>(std::vector<int>{s, r, c, k}, curand_gen);
  // auto filter = rand<T1>(std::vector<int>{k, c, kernel_d, s, r}, curand_gen);
  auto filter = rand<T1>(std::vector<int>{k, c, r, s, kernel_d}, curand_gen);
  // auto filter = rand<T1>(std::vector<int>{s, r, c, k,}, curand_gen);

  // Allocate memory for input
  auto input = rand<T1>(std::vector<int>{n, c, d, h, w}, curand_gen);

  // Allocate memory for output tensor  
  auto output = zeros<T1>(cnn.get_output_dims());

  // Warm up
  cnn.forward(input, filter, output);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < num_repeats; ++i) {
    cnn.forward(input, filter, output);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  int fwd_time = static_cast<int>(
      std::chrono::duration<double, std::micro>(end - start).count() /
      num_repeats);

  return fwd_time;
}

int main(int argc, char **argv) {
  int num_repeats = 20;

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }

  for (int dev = 0; dev < deviceCount; ++dev) {
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;

    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    std::cout
        << "d,kernel_d,w,h,c,n,k,f_w,f_h,pad_d,pad_w,pad_h,stride_d,stride_w,stride_h,fp32 time "
           "(usec),fp16 time (usec),int8 time "
           "(usec),fp16 tensor core time (usec),int8 tensor core time (usec)"
        << std::endl;

    int pad_kernels_count = 0;

    int batch = 1;

    for (const auto &problem : conv_3d) {
      // Filter parameters
      int k, c, r, s; // r - filter_h (f_h), s - filter_w (f_w)
      // Input parameters
      int n, w, h;
      // Padding
      int pad_d, pad_w, pad_h;
      // Stride
      int dstride, wstride, hstride;
      int d, kernel_d;
      std::tie(d, kernel_d, w, h, c, n, k, s, r, pad_d, pad_w, pad_h, dstride, wstride, hstride) = problem;
      n = batch;
      int fwd_time;

      std::cout << d << ",";
      std::cout << kernel_d << ",";
      std::cout << w << ",";
      std::cout << h << ",";
      std::cout << c << ",";
      std::cout << n << ",";
      std::cout << k << ",";
      std::cout << s << ",";
      std::cout << r << ",";
      std::cout << pad_d << ",";
      std::cout << pad_w << ",";
      std::cout << pad_h << ",";
      std::cout << dstride << ",";
      std::cout << wstride << ",";
      std::cout << hstride;

      // fp32 benchmark
      {
        int padded_c, padded_w, padded_h;

        padded_c = c;
        padded_h = h;
        padded_w = w;

        // fwd_time = time_cnn<float, float>(
        //     k, padded_c, r, s, n, padded_h, padded_w, pad_h, pad_w, hstride,
        //     wstride, num_repeats, curand_gen, false);
        fwd_time = time_cnn<float, float>(
          k, padded_c, r, s, n, d, kernel_d, padded_h, padded_w, pad_d, pad_h, pad_w,
          dstride, hstride, wstride, num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // fp16 benchmark
      {
        int padded_c, padded_w, padded_h;

        padded_c = c;
        padded_h = h;
        padded_w = w;

        fwd_time = time_cnn<uint16_t, uint16_t>(
            k, padded_c, r, s, n, d, kernel_d, padded_h, padded_w, pad_d, pad_h, pad_w,
            dstride, hstride, wstride, num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // int8 benchmark
      {
        int padded_c, padded_w, padded_h, padded_d;
        int pad_value;

        padded_c = c;
        padded_h = h;
        padded_w = w;
        padded_d = d;

        pad_value = 4;
        if (c % pad_value || w % pad_value || h % pad_value || d % pad_value) {
          pad_kernels_count++;
          pad_dim(padded_c, pad_value);
          pad_dim(padded_h, pad_value);
          pad_dim(padded_w, pad_value);
          pad_dim(padded_d, pad_value);
        }
        fwd_time = time_cnn<uint8_t, int>(
            k, padded_c, r, s, n, padded_d, kernel_d, padded_h, padded_w, pad_d, pad_h, pad_w, 
            dstride, hstride, wstride, num_repeats, curand_gen, false);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // fp16 tensor core benchmark
      {
        int padded_c, padded_w, padded_h;

        padded_c = c;
        padded_h = h;
        padded_w = w;

        fwd_time = time_cnn<uint16_t, uint16_t>(
            k, padded_c, r, s, n, d, kernel_d, padded_h, padded_w, pad_d, pad_h, pad_w, 
            dstride, hstride, wstride, num_repeats, curand_gen, true);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      // int8 tensor core benchmark
      {
        int padded_c, padded_w, padded_h, padded_d;
        int pad_value;

        padded_c = c;
        padded_h = h;
        padded_w = w;
        padded_d = d;

        pad_value = 4;
        if (c % pad_value || w % pad_value || h % pad_value || d % pad_value) {
          pad_kernels_count++;
          pad_dim(padded_c, pad_value);
          pad_dim(padded_h, pad_value);
          pad_dim(padded_w, pad_value);
          pad_dim(padded_d, pad_value);
        }
        fwd_time = time_cnn<uint8_t, int>(
            k, padded_c, r, s, n, padded_d, kernel_d, padded_h, padded_w, pad_d, pad_h, pad_w, 
            dstride, hstride, wstride, num_repeats, curand_gen, true);
        std::cout << "," << std::setprecision(6) << fwd_time;
      }

      std::cout << std::endl;
    }

    // Destroy all the handles
    hiprandDestroyGenerator(curand_gen);
  }

  return 0;
}